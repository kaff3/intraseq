#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include "cub/device/device_scan.cuh"

#define NUM_THREADS 256
#define TILE_SIZE 1024
#define B 4
#define get_digit(V, I) (V & (0xF << (I * 4)))
#define HISTOGRAM_SIZE 16
#define WARP 32


__global__ void kernel12(unsigned int* d_out, unsigned int* d_in, uint64_t arr_size, unsigned int* d_histogram, int curr_digit){
    __shared__ unsigned int s_tile[TILE_SIZE];
    __shared__ unsigned int s_tile_sorted[TILE_SIZE];
    __shared__ unsigned int s_histogram[HISTOGRAM_SIZE];
    int iterations = TILE_SIZE / NUM_THREADS; 
    int idx = threadIdx.x;
    int gid = blockDim.x * blockIdx.x + threadIdx.x; 

    // zero initialize histogram
    if (idx < HISTOGRAM_SIZE){
        s_histogram[idx] = 0;
    }

    for (int i = 0; i < iterations; i++){
        uint64_t arr_i = gid + i * NUM_THREADS;
        if (arr_i < arr_size) {
            unsigned int val = d_in[arr_i];
            // increment histogram
            unsigned int digit = get_digit(val, curr_digit);
            atomicAdd(s_histogram + digit, 1);
            
            // copy to shared memory
            s_tile[idx + i * NUM_THREADS] = val;
        }
    }
    __syncthreads();
    
    // copy local histogram to global memory transposed
    // Because there are 8 warps per block, if for each Warp
    // the two first threads are used, there is better
    // coalesed access.
    const unsigned int lane = idx & (WARP-1);
    if (lane == 0 || lane == 1){
        unsigned int curr_warp = idx / WARP;
        int histogram_i = curr_warp * 2 + lane;
        // p = num blocks = gridDim.x
        d_histogram[gridDim.x * histogram_i + blockIdx.x] = s_histogram[histogram_i];
    }

    // exclusive scan over histogram
    // TODO: currently sequential on a single thread
    if (idx == 0){
        for (int i = HISTOGRAM_SIZE-1; i > 0; i--){
            s_histogram[i] = s_histogram[i-1];
        }
        s_histogram[0] = 0;
        for (int i = 1; i < HISTOGRAM_SIZE; i++){
            s_histogram[i] += s_histogram[i-1];
        }
    }

    for (int i  = 0; i < iterations; i++){
        // foreach val in s_tile
        unsigned int val = s_tile[idx + i * NUM_THREADS];
        unsigned int old = atomicAdd(s_histogram + val, 1);
        s_tile_sorted[old] = val;
    }

    __syncthreads();


    // SKAL VI SKRIVE DET SORTEREDE TILBAGE?
    // KAN MAN IKKE BARE LADE DET LIGGE I SHARED OG VENTE TIL STEP 4?
    for (int i = 0; i < iterations; i++){
        d_out[gid + i * NUM_THREADS] = s_tile_sorted[idx + i * NUM_THREADS];
    }
}

// 
// __global__ void kernel3(unsigned int* d_out, unsigned int* d_in, unsigned int* d_histogram){
    
//     hipcub::DeviceScan::ExclusiveSum()

// }

__global__ void kernel4(unsigned int* d_out, unsigned int* d_in, unsigned int* d_histogram){

}

// form sorting_test.cu
void randomInitNat(unsigned int* data, const unsigned int size, const unsigned int H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

int main(int argc, char* argv[]){


    // unsigned int* vals = (unsigned int*) malloc(10000 * sizeof(unsigned int));

    const uint64_t N = atoi(argv[1]);
    // TODO: maybe check N if it is too big
    const uint64_t arr_size = N * sizeof(unsigned int);

    // Host allocations
    unsigned int* h_in  = (unsigned int*) malloc(arr_size);
    unsigned int* h_out = (unsigned int*) malloc(arr_size);

    // Create random array to sort
    randomInitNat(h_in, N, N);

    // Compute blocks and block sizes
    unsigned int num_blocks = (N + TILE_SIZE - 1) / TILE_SIZE;

    // Device allocations
    unsigned int* d_in;
    unsigned int* d_out;
    unsigned int* d_histogram;
    hipMalloc((void**)&d_in,  arr_size);
    hipMalloc((void**)&d_out, arr_size);
    hipMalloc((void**)&d_histogram, num_blocks * HISTOGRAM_SIZE);

    // Copy initial array to device
    hipMemcpy(d_in, h_in, arr_size, hipMemcpyHostToDevice);

    unsigned int* d_res;

    for (int i = 0; i < (sizeof(unsigned int)*8)/B; i++) {

        kernel12<<< num_blocks, NUM_THREADS >>>(d_out, d_in, N, d_histogram, i);

        d_res = d_out;
        // Swap input input and output
        unsigned int* tmp = d_out;
        d_out = d_in;
        d_in = tmp;
    }


    hipMemcpy(h_out, d_res, arr_size, hipMemcpyDeviceToHost);

    printf("h_out:\n");
    for (int i = 0; i < N; i++) {
        printf("%i      %i\n", h_out[i], h_in[i]);
    }


    // kernel12



    // kernel 3
    // void     *d_temp_storage = NULL;
    // size_t   temp_storage_bytes = 0;
    // hipcub::DeviceScan::ExclusiveSum();
    
    
    // kernel 4


    // Clean up memory

    return 0;
}
