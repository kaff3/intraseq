
// Include the three versions of radix we want to test
// #include "./radix.cuh"
#include"./radix-no-opt.cuh"
#include"./radix-cub.cuh"
#include"./helper.cu.h"
// Standard includes
#include<stdio.h>
#include<stdint.h>
#include<vector>
#include<sys/time.h>
#include<time.h>
#include<math.h>
#include<stdlib.h>
#include<sstream>

// Cuda includes
#include"hipcub/hipcub.hpp"
#include<hip/hip_runtime.h>


int GetMask(int b){
    int res = 0;
    for (int i = 0; i < b; i++) {
        res = res << 1;
        res = res | 1;
    }
    
    return res;
}

template<typename T>
bool validate(T* h1, T* h2, int N) {
    bool valid = true;
    for (int i = 0; i < N; i++) {
        if (h1[i] != h2[i]) {
            valid = false;
            break;
        }
    }
    return valid;
}

template<
    typename T, 
    int B, 
    int E,
    int TS >
void bench(std::vector<size_t> sizes, int gpu_runs, const char* out_file) {
    
    std::vector<float> avg_our;
    std::vector<float> avg_cub;

    for (int i = 0; i < sizes.size(); i++) {
        size_t N = sizes[i];
        printf("===============================\n");
        printf("N: %lu\n", N);
        size_t arr_size = N * sizeof(T);


        // Host allocations
        T* h_in      = (T*)malloc(arr_size);
        T* h_out_our = (T*)malloc(arr_size);
        T* h_out_cub = (T*)malloc(arr_size);

        // Instantiate our radix sort algorithm with template with a typedef
        typedef Radix<T, B, E, TS> Radix4;
        int mask = GetMask(B);

        // Device allocations
        T* d_in;
        T* d_out;
        unsigned int* d_histogram1;
        unsigned int* d_histogram2;
        unsigned int* d_histogram3;
        void*         d_tmp_storage;
        hipMalloc((void**)&d_in,  arr_size);
        hipMalloc((void**)&d_out, arr_size);
        hipMalloc((void**)&d_histogram1, Radix4::HistogramStorageSize(N));
        hipMalloc((void**)&d_histogram2, Radix4::HistogramStorageSize(N));
        hipMalloc((void**)&d_histogram3, Radix4::HistogramStorageSize(N));
        hipMalloc((void**)&d_tmp_storage, Radix4::TempStorageSize(N, d_histogram1));

        // Allocations for cub version
        void* d_tmp_storage_cub = NULL;
        size_t tmp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortKeys(d_tmp_storage_cub, tmp_storage_bytes, d_in, d_out, N);
        hipMalloc(&d_tmp_storage_cub, tmp_storage_bytes);


        // Dry runs
        Radix4::Sort(d_in, d_out, N, d_histogram1, d_histogram2, d_histogram3, d_tmp_storage, mask);
        RadixSortCub<T>(d_in, d_out, N, d_tmp_storage_cub, tmp_storage_bytes);
        hipDeviceSynchronize();

        std::vector<Timer> time_our;
        std::vector<Timer> time_cub;

        for (int j = 0; j < gpu_runs; j++) {

            // Initialize the array to be sorted and transfer to device
            randomInitNat<T>(h_in, N, N);

            // Timers for our version and cub
            Timer t1, t2;

            // Move array to device
            hipMemcpy(d_in, h_in, arr_size, hipMemcpyHostToDevice);

            // Run our version and save the result
            t1.Start();
            Radix4::Sort(d_in, d_out, N, d_histogram1, d_histogram2, d_histogram3, d_tmp_storage, mask);
            hipDeviceSynchronize();
            t1.Stop();

            #ifdef RADIX_VALIDATE
            // Save sorted array to host for validation
            hipMemcpy(h_out_our, d_in, arr_size, hipMemcpyDeviceToHost);
            #endif

            // Now the CUB version
            hipMemcpy(d_in, h_in, arr_size, hipMemcpyHostToDevice);
 
            t2.Start();
            RadixSortCub<T>(d_in, d_out, N, d_tmp_storage_cub, tmp_storage_bytes);
            hipDeviceSynchronize();
            t2.Stop();


            #ifdef RADIX_VALIDATE
            hipMemcpy(h_out_cub, d_out, arr_size, hipMemcpyDeviceToHost);
            // Print if we do not validate
            if (!validate<T>(h_out_our, h_out_cub, N)) {
                printf("INVALID. Size %i run %i\n", N, j);
            }
            #endif

            // Save runtimes
            time_our.push_back(t1);
            time_cub.push_back(t2);
        }

        // Save the average runtimes
        float run_our = average(time_our);
        float run_cub = average(time_cub);

        avg_our.push_back(run_our);
        avg_cub.push_back(run_cub);

        printf("Our: %.2f\n", run_our);
        printf("Cub: %.2f\n", run_cub);
        printf("factor: %f\n", run_our/run_cub);


        // Have to allocate and free each iteration of outer loop as the sizes change but they are not timed
        hipFree(d_in);
        hipFree(d_out);
        hipFree(d_histogram1);
        hipFree(d_histogram2);
        hipFree(d_histogram3);
        hipFree(d_tmp_storage);
        hipFree(d_tmp_storage_cub);
        free(h_in);
        free(h_out_our);
        free(h_out_cub);
    }

    writeRuntimes(sizes, avg_our, avg_cub, out_file);

}

template<
    typename T, 
    int B, 
    int E,
    int TS >
void benchTuning(std::vector<size_t> sizes, int gpu_runs, const char* out_file) {

    std::vector<float> avg_times;
    
    for (int i = 0; i < sizes.size(); i++) {
        size_t N = sizes[i];
        printf("===============================\n");
        printf("N: %lu\n", N);
        size_t arr_size = N * sizeof(T);


        // Host allocations
        T* h_in      = (T*)malloc(arr_size);

        // Instantiate our radix sort algorithm with template with a typedef
        typedef Radix<T, B, E, TS> Radix4;
        int mask = GetMask(B);

        // Device allocations
        T* d_in;
        T* d_out;
        unsigned int* d_histogram1;
        unsigned int* d_histogram2;
        unsigned int* d_histogram3;
        void*         d_tmp_storage;
        hipMalloc((void**)&d_in,  arr_size);
        hipMalloc((void**)&d_out, arr_size);
        hipMalloc((void**)&d_histogram1, Radix4::HistogramStorageSize(N));
        hipMalloc((void**)&d_histogram2, Radix4::HistogramStorageSize(N));
        hipMalloc((void**)&d_histogram3, Radix4::HistogramStorageSize(N));
        hipMalloc((void**)&d_tmp_storage, Radix4::TempStorageSize(N, d_histogram1));

        // Dry run
        Radix4::Sort(d_in, d_out, N, d_histogram1, d_histogram2, d_histogram3, d_tmp_storage, mask);
        hipDeviceSynchronize();

        std::vector<Timer> times;

        for (int j = 0; j < gpu_runs; j++) {

            Timer t1;

             // Initialize the array to be sorted and transfer to device
            randomInitNat<T>(h_in, N, N);

            // Move array to device
            hipMemcpy(d_in, h_in, arr_size, hipMemcpyHostToDevice);

            // Run our version and save the result
            t1.Start();
            Radix4::Sort(d_in, d_out, N, d_histogram1, d_histogram2, d_histogram3, d_tmp_storage, mask);
            hipDeviceSynchronize();
            t1.Stop();

            times.push_back(t1);
        }

        float avg = average(times);
        avg_times.push_back(avg);

        printf("Time: %.2f\n", avg);

        hipFree(d_in);
        hipFree(d_out);
        hipFree(d_histogram1);
        hipFree(d_histogram2);
        hipFree(d_histogram3);
        hipFree(d_tmp_storage);
        free(h_in);
    }

    writeRuntimes(sizes, avg_times, avg_times, out_file);

}


int main(int argc, char* argv[]) {

    if (argc < 2) {
        printf("Usage: ./radix <gpu runs>\n");
        return 0;
    }

    int gpu_runs = atoi(argv[1]);

    std::vector<size_t> sizes;
    sizes.push_back(100000);
    sizes.push_back(250000);
    sizes.push_back(500000);
    sizes.push_back(750000);
    sizes.push_back(1000000);
    sizes.push_back(2500000);
    sizes.push_back(5000000);
    sizes.push_back(7500000);
    sizes.push_back(10000000);
    sizes.push_back(25000000);
    sizes.push_back(50000000);
    sizes.push_back(75000000);
    sizes.push_back(100000000);
    sizes.push_back(250000000);
    sizes.push_back(500000000);
    sizes.push_back(750000000);

    // printf("===== Parameter B tests =====\n");
    // benchTuning<unsigned int, 1, 4, 256>(sizes, gpu_runs, "data/B-1-4-256.csv");
    // benchTuning<unsigned int, 4, 4, 256>(sizes, gpu_runs, "data/B-4-4-256.csv");
    // benchTuning<unsigned int, 8, 4, 256>(sizes, gpu_runs, "data/B-8-4-256.csv");

    // printf("===== Parameter E tests =====\n");
    // benchTuning<unsigned int, 4, 1, 256>(sizes, gpu_runs, "data/E-4-1-256.csv");
    benchTuning<unsigned int, 4, 4, 256>(sizes, gpu_runs, "data/E-4-4-256.csv");
    // benchTuning<unsigned int, 4, 8, 256>(sizes, gpu_runs, "data/E-4-8-256.csv");

    // printf("===== Parameter TS tests =====\n");
    // benchTuning<unsigned int, 4, 4, 256*1>(sizes, gpu_runs, "data/TS-4-4-256.csv");
    // benchTuning<unsigned int, 4, 4, 256*2>(sizes, gpu_runs, "data/TS-4-4-512.csv");
    // benchTuning<unsigned int, 4, 4, 256*3>(sizes, gpu_runs, "data/TS-4-4-768.csv");
    // benchTuning<unsigned int, 4, 4, 256*4>(sizes, gpu_runs, "data/TS-4-4-1024.csv");


    // printf("\nUnsigned int:\n");
    // bench<unsigned int, 8, 4, 512>(sizes, gpu_runs, "data/u32-8-4-512.csv");

    // printf("\nUnsigned long:\n");
    // bench<unsigned long, 8, 4, 512>(sizes, gpu_runs, "data/u64-8-4-512.csv");

    // printf("\nUnsigned short:\n");
    // bench<unsigned short, 8, 4, 512>(sizes, gpu_runs, "data/u16-8-4-512.csv");
    
    // printf("\nUnsigned char:\n");
    // bench<unsigned char, 8, 4, 512>(sizes, gpu_runs, "data/u8-8-4-512.csv");


    return 0;
}





