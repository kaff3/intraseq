#include<stdio.h>
#include<stdlib.h>
#include"./scan.cuh"
#include"../timing.h"
#include<hip/hip_runtime.h>
#include<algorithm>
#include<iterator>

#define TEST_SIZE (1024 * 1500)
#define BLOCK_SIZE 1024


// void testReduce() {
//     int foo[REDUCE_TEST_SIZE];
//     std::fill(std::begin(foo), std::end(foo), 1);
//     int num_blocks = (REDUCE_TEST_SIZE + REDUCE_TEST_BLOCK_SIZE - 1) / REDUCE_TEST_BLOCK_SIZE;
//     int shmem_size = REDUCE_TEST_BLOCK_SIZE * sizeof(int);

//     printf("num blocks = %i\n", num_blocks);
//     printf("shmem size = %i bytes\n", shmem_size);
    
//     int* d_in;
//     int* d_out;
//     hipMalloc((void**)&d_in, REDUCE_TEST_SIZE * sizeof(int));
//     hipMalloc((void**)&d_out, num_blocks * sizeof(int));

//     printf("Copying to device\n");
//     hipMemcpy(d_in, (void*)&foo, REDUCE_TEST_SIZE * sizeof(int), hipMemcpyHostToDevice);

//     printf("Running kernel\n");
//     reduce_kernel<int><<<num_blocks, REDUCE_TEST_BLOCK_SIZE, shmem_size>>>(d_in, d_out, REDUCE_TEST_SIZE);

//     printf("Copying to host\n");
//     hipMemcpy((void*)&foo, d_out, num_blocks * sizeof(int), hipMemcpyDeviceToHost);

//     printf("Running loop \n");
//     int sum = 0;
//     for (int i = 0; i < num_blocks; i++) {
//         sum += foo[i];
//     }

//     hipDeviceSynchronize();
//     printf("Reduced: %i\n", sum);

//     hipFree((void*)d_in);
//     hipFree((void*)d_out);
// }


template<typename T>
void intraBlockScanBench() {
    int num_blocks = TEST_SIZE / BLOCK_SIZE;

    // Create the array to be scanned
    T* arr1 = (T*)malloc(TEST_SIZE*sizeof(T));
    T* arr2 = (T*)malloc(TEST_SIZE*sizeof(T));
    // std::fill(std::begin(arr1), std::end(arr1), 1);
    for (size_t i = 0; i < TEST_SIZE; i++) {
        arr1[i] = 1;
    }


    // Create device memory and copy
    T* d_in1;
    T* d_in2;
    hipMalloc((void**)&d_in1, TEST_SIZE*sizeof(T));
    hipMalloc((void**)&d_in2, TEST_SIZE*sizeof(T));
    hipMemcpy(d_in1, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_in2, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Dry run
    // scan_kernel<T><<<num_blocks, BLOCK_SIZE>>>(d_in1, TEST_SIZE, 1000);
    // hipDeviceSynchronize();
    // hipMemcpy(d_in1, (void*)&arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Benchamrking
    Timer t1, t2;
    
    const size_t iterations = 1000000;

    t1.Start();
    scan_kernel<T><<<num_blocks, BLOCK_SIZE>>>(d_in1, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t1.Stop();

    t2.Start();
    scan_kernel_seq<T><<<num_blocks, BLOCK_SIZE/4>>>(d_in2, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t2.Stop();

    printf("nor = %.2f\n", t1.Get());
    printf("seq = %.2f\n", t2.Get());

    hipMemcpy((void*)arr1, d_in1, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy((void*)arr2, d_in2, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);

    printf("loop starting\n");
    bool succes = true;
    for (size_t i = 0; i < TEST_SIZE; i++) {
       if (arr1[i] != arr2[i]) {
           succes = false;
           printf("oh no at i=%i\n", i);
           break;
       }
    }

    printf("succes = %i\n", succes);

    hipFree(d_in1);
    hipFree(d_in2);
}



int main(int argc, char* argv[]) {
    intraBlockScanBench<unsigned int>();
    return 0;
}