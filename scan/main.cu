#include<stdio.h>
#include<stdlib.h>
#include"./scan.cuh"
#include"../timing.h"
#include<hip/hip_runtime.h>
#include<algorithm>
#include<iterator>

#define TEST_SIZE (1024 * 1)
#define BLOCK_SIZE 1024

template<typename T>
void intraBlockScanBench() {
    int num_blocks = TEST_SIZE / BLOCK_SIZE;

    // Create the array to be scanned
    T* arr1 = (T*)malloc(TEST_SIZE*sizeof(T));
    T* arr2 = (T*)malloc(TEST_SIZE*sizeof(T));
    // std::fill(std::begin(arr1), std::end(arr1), 1);
    for (size_t i = 0; i < TEST_SIZE; i++) {
        arr1[i] = 1;
    }


    // Create device memory and copy
    T* d_in1;
    T* d_in2;
    hipMalloc((void**)&d_in1, TEST_SIZE*sizeof(T));
    hipMalloc((void**)&d_in2, TEST_SIZE*sizeof(T));
    hipMemcpy(d_in1, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_in2, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Dry run
    // scan_kernel<T><<<num_blocks, BLOCK_SIZE>>>(d_in1, TEST_SIZE, 1000);
    // hipDeviceSynchronize();
    // hipMemcpy(d_in1, (void*)&arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Benchamrking
    Timer t1, t2;
    
    const size_t iterations = 1;

    t1.Start();
    scan_kernel<T><<<num_blocks, BLOCK_SIZE>>>(d_in1, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t1.Stop();

    t2.Start();
    scan_kernel_seq<T><<<num_blocks, BLOCK_SIZE/4>>>(d_in2, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t2.Stop();

    printf("nor = %.2f\n", t1.Get());
    printf("seq = %.2f\n", t2.Get());

    hipMemcpy((void*)arr1, d_in1, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy((void*)arr2, d_in2, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);

    printf("loop starting\n");
    bool succes = true;
    for (size_t i = 0; i < TEST_SIZE; i++) {
       if (arr1[i] != arr2[i]) {
           succes = false;
           printf("oh no at i=%i\n", i);
           printf("Arr1: %u\n", arr1[i]);
           printf("Arr2: %u\n", arr2[i]);
           break;
       }
    }

    printf("succes = %i\n", succes);

    hipFree(d_in1);
    hipFree(d_in2);
}



int main(int argc, char* argv[]) {
    intraBlockScanBench<unsigned int>();
    return 0;
}