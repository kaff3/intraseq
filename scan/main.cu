#include<stdio.h>
#include<stdlib.h>
#include"./scan.cuh"
#include"../shared/timing.h"
#include<hip/hip_runtime.h>
#include<algorithm>
#include<iterator>

// #define TEST_SIZE (1024 * 1000)
// #define BLOCK_SIZE 1024

template<typename T>
void intraBlockScanBench(const unsigned int block_size, 
                         const unsigned int num_blocks,
                         const unsigned int num_elems,
                         const unsigned int iterations) 
{
    
    // compute total number of elements
    const unsigned int TEST_SIZE = block_size * num_blocks;

    // Create the array to be scanned
    T* arr1 = (T*)malloc(TEST_SIZE*sizeof(T));
    T* arr2 = (T*)malloc(TEST_SIZE*sizeof(T));
    T* arr3 = (T*)malloc(TEST_SIZE*sizeof(T));
    for (size_t i = 0; i < TEST_SIZE; i++) {
        arr1[i] = 1;
    }


    // Create device memory and copy
    T* d_in1;
    T* d_in2;
    T* d_in3;
    hipMalloc((void**)&d_in1, TEST_SIZE*sizeof(T));
    hipMalloc((void**)&d_in2, TEST_SIZE*sizeof(T));
    hipMalloc((void**)&d_in3, TEST_SIZE*sizeof(T));
    hipMemcpy(d_in1, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_in2, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_in3, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Dry run
    scan_kernel<T><<<num_blocks, block_size>>>(d_in1, TEST_SIZE, 100);
    hipDeviceSynchronize();
    hipMemcpy(d_in1, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Benchmarking
    Timer t1, t2, t3;

    t1.Start();
    scan_kernel<T><<<num_blocks, block_size>>>(d_in1, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t1.Stop();

    t2.Start();
    scan_kernel_seq<T><<<num_blocks, block_size/num_elems>>>(d_in2, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t2.Stop();

    t3.Start();
    scan_kernel_seq_reg<T><<<num_blocks, block_size/num_elems>>>(d_in3, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t3.Stop();

    printf("nor = %.2f\n", t1.Get());
    printf("seq = %.2f\n", t2.Get());
    printf("reg = %.2f\n", t3.Get());

    hipMemcpy((void*)arr1, d_in1, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy((void*)arr2, d_in2, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy((void*)arr3, d_in3, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);

    printf("loop starting\n");
    bool success = true;
    for (size_t i = 0; i < TEST_SIZE; i++) {
       if ( (arr1[i] != arr2[i]) || (arr2[i] != arr3[i])) {
           printf("Arr1: %u\n", arr1[i]);
           printf("Arr2: %u\n", arr2[i]);
           printf("Arr3: %u\n", arr3[i]);
           success = false;
           printf("oh no at i=%i\n", i);
           break;
       }
    }

    printf("success = %i\n", success);

    hipFree(d_in1);
    hipFree(d_in2);
    hipFree(d_in3);
    free(arr1);
    free(arr2);
    free(arr3);
}



int main(int argc, char* argv[]) {
    const size_t iterations = 1000000;

    intraBlockScanBench<unsigned int>(1024, 1000, 4, iterations);
    return 0;
}
