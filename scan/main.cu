#include<stdio.h>
#include<stdlib.h>
#include"./scan.cuh"
#include"../shared/timing.h"
#include<hip/hip_runtime.h>
#include<algorithm>
#include<iterator>

#define TEST_SIZE (1024 * 2000)
#define BLOCK_SIZE 1024

template<typename T>
void intraBlockScanBench() {
    int num_blocks = TEST_SIZE / BLOCK_SIZE;

    // Create the array to be scanned
    T* arr1 = (T*)malloc(TEST_SIZE*sizeof(T));
    T* arr2 = (T*)malloc(TEST_SIZE*sizeof(T));
    T* arr3 = (T*)malloc(TEST_SIZE*sizeof(T));
    for (size_t i = 0; i < TEST_SIZE; i++) {
        arr1[i] = 1;
    }


    // Create device memory and copy
    T* d_in1;
    T* d_in2;
    T* d_in3;
    hipMalloc((void**)&d_in1, TEST_SIZE*sizeof(T));
    hipMalloc((void**)&d_in2, TEST_SIZE*sizeof(T));
    hipMalloc((void**)&d_in3, TEST_SIZE*sizeof(T));
    hipMemcpy(d_in1, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_in2, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_in3, (void*)arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Dry run
    //scan_kernel<T><<<num_blocks, BLOCK_SIZE>>>(d_in1, TEST_SIZE, 100);
    //hipDeviceSynchronize();
    //hipMemcpy(d_in1, (void*)&arr1, TEST_SIZE*sizeof(T), hipMemcpyHostToDevice);

    // Benchmarking
    Timer t1, t2, t3;
    
    const size_t iterations = 5000000;

    t1.Start();
    scan_kernel<T><<<num_blocks, BLOCK_SIZE>>>(d_in1, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t1.Stop();

    t2.Start();
    scan_kernel_seq<T><<<num_blocks, BLOCK_SIZE/4>>>(d_in2, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t2.Stop();

    t3.Start();
    scan_kernel_seq_reg<T><<<num_blocks, BLOCK_SIZE/4>>>(d_in3, TEST_SIZE, iterations);
    hipDeviceSynchronize();
    t3.Stop();

    printf("nor = %.2f\n", t1.Get());
    printf("seq = %.2f\n", t2.Get());
    printf("reg = %.2f\n", t3.Get());

    hipMemcpy((void*)arr1, d_in1, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy((void*)arr2, d_in2, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);
    hipMemcpy((void*)arr3, d_in3, TEST_SIZE*sizeof(T), hipMemcpyDeviceToHost);

    printf("loop starting\n");
    bool success = true;
    for (size_t i = 0; i < TEST_SIZE; i++) {
       if ( (arr1[i] != arr2[i]) || (arr2[i] != arr3[i])) {
           printf("Arr1: %u\n", arr1[i]);
           printf("Arr2: %u\n", arr2[i]);
           printf("Arr3: %u\n", arr3[i]);
           success = false;
           printf("oh no at i=%i\n", i);
           break;
       }
    }

    printf("success = %i\n", success);

    hipFree(d_in1);
    hipFree(d_in2);
    hipFree(d_in3);
    free(arr1);
    free(arr2);
    free(arr3);
}



int main(int argc, char* argv[]) {
    intraBlockScanBench<unsigned int>();
    return 0;
}
