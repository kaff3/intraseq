#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_cooperative_groups.h>
#include <stdint.h>

#include "hipcub/hipcub.hpp"


#define THREAD_ELEMENTS     4   // 
#define NUM_THREADS         256
#define TILE_SIZE           (NUM_THREADS * THREAD_ELEMENTS)
#define B                   4
#define HISTOGRAM_SIZE      16  // 2^B
#define GET_DIGIT(V, I, M)  ((V >> I) & M)

// #define NUM_POSITIONS       (NUM_THREADS * 2)


__global__ void kernel12(unsigned int* d_out, unsigned int* d_in, uint64_t arr_size, unsigned int* d_histogram, int curr_digit){

    __shared__ unsigned int s_tile[TILE_SIZE];

    // Load to shared memory
    #pragma unroll
    for (int i = 0; i < THREAD_ELEMENTS; i++) {
        unsigned int s_index = threadIdx.x + i * blockDim.x;
        unsigned int d_index = blockIdx.x * TILE_SIZE + s_index;
        if (d_index < arr_size) {
            s_tile[s_index] = d_in[d_index];
        }
    }
    __syncthreads();


    // Sort in shared memory b iterations 1-bit split
    unsigned int elements[THREAD_ELEMENTS];
    for (int i = 0; i < B; i++) {

        // Read elements
        unsigned int index;
        #pragma unroll
        for (int j = 0; j < THREAD_ELEMENTS; j++) {
            index = threadIdx.x * THREAD_ELEMENTS + j;
            if (index < arr_size) {
                elements[j] = s_tile[index];
            }
        }
        __syncthreads();

        unsigned int ps0 = 0;
        unsigned int ps1 = 0;

        #pragma unroll
        for (int j = 0; j < THREAD_ELEMENTS; j++) {
            unsigned int index = threadIdx.x * THREAD_ELEMENTS + j;
            if (index < arr_size) {
                unsigned int bit = GET_DIGIT(elements[j], curr_digit*B+i, 0x1);
                ps0 += (bit == 0 ? 1 : 0);
                ps1 += (bit == 1 ? 1 : 0);
            }
        }
        __syncthreads();


        // ======== OLD ==========
        // static const int num_positions = 2 * NUM_THREADS;
        // __shared__ unsigned int positions[num_positions];

        // // Write positions to shared memory to prepare for scan
        // positions[threadIdx.x]              = ps0;
        // positions[blockDim.x + threadIdx.x] = ps1;
        // __syncthreads();

        // Perform scan. TODO: Make it not sequential
        // if (threadIdx.x == 0) {
        //     for (int j = 1; j < num_positions; j++) {
        //         positions[j] += positions[j-1];
        //     }
        // }

        // ps0 = (threadIdx.x == 0 ? 0 : positions[threadIdx.x - 1]);
        // ps1 = positions[blockDim.x + threadIdx.x - 1];

        // ========= NEW ============
        // Perform a scan across threads
        typedef hipcub::BlockScan<unsigned int, NUM_THREADS> BlockScan;

        __shared__ typename BlockScan::TempStorage ps0_storage;
        __shared__ typename BlockScan::TempStorage ps1_storage;
        __shared__ unsigned int aggregate;

        BlockScan(ps0_storage).ExclusiveScan(ps0, ps0, 0, hipcub::Sum(), aggregate);
        __syncthreads();
        BlockScan(ps1_storage).ExclusiveScan(ps1, ps1, aggregate, hipcub::Sum());
        __syncthreads();



        // Sort by scattering
        #pragma unroll
        for (int j = 0; j < THREAD_ELEMENTS; j++) {
            unsigned int index = threadIdx.x * THREAD_ELEMENTS + j;
            if (index < arr_size) {
                unsigned int bit = GET_DIGIT(elements[j], curr_digit*B+i, 0x1);
                unsigned int pos = (bit == 0 ? ps0 : ps1);
                ps0 += (bit == 0 ? 1 : 0);
                ps1 += (bit == 1 ? 1 : 0);
                s_tile[pos] = elements[j];
            }
        }
        __syncthreads(); // For next iteration

    } // Big loop end



    // zero initialize histogram
    __shared__ unsigned int s_histogram[HISTOGRAM_SIZE];
    if (threadIdx.x < HISTOGRAM_SIZE){
        s_histogram[threadIdx.x] = 0;
    }
    __syncthreads();


    // Compute final histogram
    for (int i = 0; i < THREAD_ELEMENTS; i++) {
        unsigned int index = threadIdx.x * THREAD_ELEMENTS + i;
        if (index < arr_size) {
            unsigned int digit = GET_DIGIT(elements[i], curr_digit*B+1, 0xF); // TODO: Fix mask somehow
            atomicAdd(s_histogram + digit, 1);
        }
    }
    __syncthreads();

    // Write histogram to global memory
    if (threadIdx.x < HISTOGRAM_SIZE) {
        d_histogram[gridDim.x * threadIdx.x + blockIdx.x] = s_histogram[threadIdx.x];
    }

    // Write sorted tile back to global memory. coalesced
    for (int i = 0; i < THREAD_ELEMENTS; i++) {
        unsigned int s_index = threadIdx.x + blockDim.x * i;
        unsigned int d_index = blockIdx.x * TILE_SIZE + s_index;
        if (d_index < arr_size) {
            d_out[d_index] = s_tile[s_index];
        }
    }
}



// form sorting_test.cu
void randomInitNat(unsigned int* data, const unsigned int size, const unsigned int H) {
    for (int i = 0; i < size; ++i) {
        unsigned long int r = rand();
        data[i] = r % H;
    }
}

int main(int argc, char* argv[]){

    const uint64_t N = atoi(argv[1]);
    // TODO: maybe check N if it is too big
    const uint64_t arr_size = N * sizeof(unsigned int);

    // Host allocations
    unsigned int* h_in  = (unsigned int*) malloc(arr_size);
    unsigned int* h_out = (unsigned int*) malloc(arr_size);

    // Create random array to sort
    randomInitNat(h_in, N, N);

    // Compute blocks and block sizes
    unsigned int num_blocks = (N + TILE_SIZE - 1) / TILE_SIZE;

    // Device allocations
    unsigned int* d_in;
    unsigned int* d_out;
    unsigned int* d_histogram;
    hipMalloc((void**)&d_in,  arr_size);
    hipMalloc((void**)&d_out, arr_size);
    hipMalloc((void**)&d_histogram, num_blocks * HISTOGRAM_SIZE * sizeof(unsigned int));

    // Copy initial array to device
    hipMemcpy(d_in, h_in, arr_size, hipMemcpyHostToDevice);


    printf("num blocks:  %i\n", num_blocks);
    printf("num threads: %i\n", num_blocks*NUM_THREADS);
    printf("tile size:   %i\n", TILE_SIZE);


    for (int i = 0; i < (sizeof(unsigned int)*8)/B; i++) {
        kernel12<<< num_blocks, NUM_THREADS >>>(d_out, d_in, N, d_histogram, i);

        // Swap input input and output
        unsigned int* tmp;
        tmp = d_in;
        d_in  = d_out;
        d_out = tmp;
    }


    // Copy from device to print result
    hipMemcpy(h_out, d_out, arr_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        printf("%10x      %10x\n", h_out[i], h_in[i]);
    }



    // kernel12



    // kernel 3
    // void     *d_temp_storage = NULL;
    // size_t   temp_storage_bytes = 0;
    // hipcub::DeviceScan::ExclusiveSum();
    
    
    // kernel 4


    // Clean up memory

    return 0;
}
